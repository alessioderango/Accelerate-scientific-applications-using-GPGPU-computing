#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "util.hpp"

// ----------------------------------------------------------------------------
// I/O parameters used to index argv[]
// ----------------------------------------------------------------------------
#define HEADER_PATH_ID 1
#define DEM_PATH_ID 2
#define SOURCE_PATH_ID 3
#define OUTPUT_PATH_ID 4
#define STEPS_ID 5
#define BLOCK_SIZE_X 6
#define BLOCK_SIZE_Y 7
// ----------------------------------------------------------------------------
// Simulation parameters
// ----------------------------------------------------------------------------
#define P_R 0.5
#define P_EPSILON 0.001
#define ADJACENT_CELLS 4
#define STRLEN 256

// ----------------------------------------------------------------------------
// Read/Write access macros linearizing single/multy layer buffer 2D indices
// ----------------------------------------------------------------------------
#define SET(M, columns, i, j, value) ((M)[(((i) * (columns)) + (j))] = (value))
#define GET(M, columns, i, j) (M[(((i) * (columns)) + (j))])
#define BUF_SET(M, rows, columns, n, i, j, value) ( (M)[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] = (value) )
#define BUF_GET(M, rows, columns, n, i, j) ( M[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] )

// ----------------------------------------------------------------------------
// I/O functions
// ----------------------------------------------------------------------------
void readHeaderInfo(char* path, int &nrows, int &ncols, /*double &xllcorner, double &yllcorner, double &cellsize,*/ double &nodata)
{
	FILE* f;

	if ( (f = fopen(path,"r") ) == 0){
		printf("%s configuration header file not found\n", path);
		exit(0);
	}

	//Reading the header
	char str[STRLEN];
	fscanf(f,"%s",&str); fscanf(f,"%s",&str); ncols = atoi(str);      //ncols
	fscanf(f,"%s",&str); fscanf(f,"%s",&str); nrows = atoi(str);      //nrows
	fscanf(f,"%s",&str); fscanf(f,"%s",&str); //xllcorner = atof(str);  //xllcorner
	fscanf(f,"%s",&str); fscanf(f,"%s",&str); //yllcorner = atof(str);  //yllcorner
	fscanf(f,"%s",&str); fscanf(f,"%s",&str); //cellsize = atof(str);   //cellsize
	fscanf(f,"%s",&str); fscanf(f,"%s",&str); nodata = atof(str);     //NODATA_value
}

bool loadGrid2D(double *M, int rows, int columns, char *path)
{
	FILE *f = fopen(path, "r");

	if (!f) {
		printf("%s grid file not found\n", path);
		exit(0);
	}

	char str[STRLEN];
	for (int i = 0; i < rows; i++)
		for (int j = 0; j < columns; j++)
		{
			fscanf(f, "%s", str);
			SET(M, columns, i, j, atof(str));
		}

	fclose(f);

	return true;
}

bool saveGrid2Dr(double *M, int rows, int columns, char *path)
{
	FILE *f;
	f = fopen(path, "w");

	if (!f)
		return false;

	char str[STRLEN];
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < columns; j++)
		{
			sprintf(str, "%f ", GET(M, columns, i, j));
			fprintf(f, "%s ", str);
		}
		fprintf(f, "\n");
	}

	fclose(f);

	return true;
}

double* addLayer2D(int rows, int columns)
{
	double *tmp = (double *)malloc(sizeof(double) * rows * columns);
	if (!tmp)
		return NULL;
	return tmp;
}

// ----------------------------------------------------------------------------
// init kernel, called once before the simulation loop
// ----------------------------------------------------------------------------
void sciddicaTSimulationInit(int i, int j, int r, int c, double* Sz, double* Sh)
{
	double z, h;
	h = GET(Sh, c, i, j);

	if (h > 0.0)
	{
		z = GET(Sz, c, i, j);
		SET(Sz, c, i, j, z - h);
	}
}

// ----------------------------------------------------------------------------
// computing kernels, aka elementary processes in the XCA terminology
// ----------------------------------------------------------------------------
__global__ void sciddicaTResetFlows(int r, int c, double nodata, double* Sf)
{
	int j = (blockIdx.x * blockDim.x + threadIdx.x)+1;
	int i = (blockIdx.y * blockDim.y + threadIdx.y)+1;
	if(i < r && j < c)
	{
		BUF_SET(Sf, r, c, 0, i, j, 0.0);
		BUF_SET(Sf, r, c, 1, i, j, 0.0);
		BUF_SET(Sf, r, c, 2, i, j, 0.0);
		BUF_SET(Sf, r, c, 3, i, j, 0.0);
	}
}

__global__ void sciddicaTFlowsComputation(int r,
		int c,
		double nodata,
		double *Sz,
		double *Sh,
		double *Sf,
		double p_r,
		double p_epsilon)
{
	int j = (blockIdx.x * blockDim.x + threadIdx.x)+1;
	int i = (blockIdx.y * blockDim.y + threadIdx.y)+1;

	if(i < r && j < c)
	{
		int Xi[] = {0, -1,  0,  0,  1};// Xj: von Neuman neighborhood row coordinates (see below)
		int Xj[] = {0,  0, -1,  1,  0};// Xj: von Neuman neighborhood col coordinates (see below)

		bool eliminated_cells[5] = {false, false, false, false, false};
		bool again;
		int cells_count;
		double average;
		double m;
		double u[5];
		int n;
		double z, h;

		m = GET(Sh, c, i, j) - p_epsilon;
		u[0] = GET(Sz, c, i, j) + p_epsilon;
		z = GET(Sz, c, i + Xi[1], j + Xj[1]);
		h = GET(Sh, c, i + Xi[1], j + Xj[1]);
		u[1] = z + h;
		z = GET(Sz, c, i + Xi[2], j + Xj[2]);
		h = GET(Sh, c, i + Xi[2], j + Xj[2]);
		u[2] = z + h;
		z = GET(Sz, c, i + Xi[3], j + Xj[3]);
		h = GET(Sh, c, i + Xi[3], j + Xj[3]);
		u[3] = z + h;
		z = GET(Sz, c, i + Xi[4], j + Xj[4]);
		h = GET(Sh, c, i + Xi[4], j + Xj[4]);
		u[4] = z + h;

		do
		{
			again = false;
			average = m;
			cells_count = 0;

			for (n = 0; n < 5; n++)
				if (!eliminated_cells[n])
				{
					average += u[n];
					cells_count++;
				}

			if (cells_count != 0)
				average /= cells_count;

			for (n = 0; n < 5; n++)
				if ((average <= u[n]) && (!eliminated_cells[n]))
				{
					eliminated_cells[n] = true;
					again = true;
				}
		} while (again);

		if (!eliminated_cells[1]) BUF_SET(Sf, r, c, 0, i, j, (average - u[1]) * p_r);
		if (!eliminated_cells[2]) BUF_SET(Sf, r, c, 1, i, j, (average - u[2]) * p_r);
		if (!eliminated_cells[3]) BUF_SET(Sf, r, c, 2, i, j, (average - u[3]) * p_r);
		if (!eliminated_cells[4]) BUF_SET(Sf, r, c, 3, i, j, (average - u[4]) * p_r);
	}
}

__global__ void sciddicaTWidthUpdate(int r,
		int c,
		double nodata,
		double *Sz,
		double *Sh,
		double *Sf)
{
	int j = (blockIdx.x * blockDim.x + threadIdx.x)+1;
	int i = (blockIdx.y * blockDim.y + threadIdx.y)+1;

	int Xi[] = {0, -1,  0,  0,  1};// Xj: von Neuman neighborhood row coordinates (see below)
	int Xj[] = {0,  0, -1,  1,  0};// Xj: von Neuman neighborhood col coordinates (see below)
	if(i < r && j < c)
	{
		double h_next;
		h_next = GET(Sh, c, i, j);
		h_next += BUF_GET(Sf, r, c, 3, i+Xi[1], j+Xj[1]) - BUF_GET(Sf, r, c, 0, i, j);
		h_next += BUF_GET(Sf, r, c, 2, i+Xi[2], j+Xj[2]) - BUF_GET(Sf, r, c, 1, i, j);
		h_next += BUF_GET(Sf, r, c, 1, i+Xi[3], j+Xj[3]) - BUF_GET(Sf, r, c, 2, i, j);
		h_next += BUF_GET(Sf, r, c, 0, i+Xi[4], j+Xj[4]) - BUF_GET(Sf, r, c, 3, i, j);
		SET(Sh, c, i, j, h_next);
	}
}

// ----------------------------------------------------------------------------
// Function main()
// ----------------------------------------------------------------------------
int main(int argc, char **argv)
{
	int rows, cols;
	double nodata;
	readHeaderInfo(argv[HEADER_PATH_ID], rows, cols, nodata);

	int r = rows;                  // r: grid rows
	int c = cols;                  // c: grid columns
	int i_start = 1, i_end = r-1;  // [i_start,i_end[: kernels application range along the rows
	int j_start = 1, j_end = c-1;  // [i_start,i_end[: kernels application range along the rows
	double *Sz;                    // Sz: substate (grid) containing the cells' altitude a.s.l.
	double *Sh;                    // Sh: substate (grid) containing the cells' flow thickness
	double *Sf;                    // Sf: 4 substates containing the flows towards the 4 neighs
				       //  int Xi[] = {0, -1,  0,  0,  1};// Xj: von Neuman neighborhood row coordinates (see below)
				       //  int Xj[] = {0,  0, -1,  1,  0};// Xj: von Neuman neighborhood col coordinates (see below)
	double p_r = P_R;              // p_r: minimization algorithm outflows dumping factor
	double p_epsilon = P_EPSILON;  // p_epsilon: frictional parameter threshold
	int steps = atoi(argv[STEPS_ID]); //steps: simulation steps

	int bs_x, bs_y; //block_size.x and block_size.y

	char *e = argv[BLOCK_SIZE_X];
	bs_x = atoi(e);
	e = argv[BLOCK_SIZE_Y];
	bs_y = atoi(e);

	// The adopted von Neuman neighborhood
	// Format: flow_index:cell_label:(row_index,col_index)
	//
	//   cell_label in [0,1,2,3,4]: label assigned to each cell in the neighborhood
	//   flow_index in   [0,1,2,3]: outgoing flow indices in Sf from cell 0 to the others
	//       (row_index,col_index): 2D relative indices of the cells
	//
	//               |0:1:(-1, 0)|
	//   |1:2:( 0,-1)| :0:( 0, 0)|2:3:( 0, 1)|
	//               |3:4:( 1, 0)|
	//
	//

	Sz = addLayer2D(r, c);                 // Allocates the Sz substate grid
	Sh = addLayer2D(r, c);                 // Allocates the Sh substate grid
	Sf = addLayer2D(ADJACENT_CELLS* r, c); // Allocates the Sf substates grid,
					       //   having one layer for each adjacent cell

	loadGrid2D(Sz, r, c, argv[DEM_PATH_ID]);   // Load Sz from file
	loadGrid2D(Sh, r, c, argv[SOURCE_PATH_ID]);// Load Sh from file

	for (int i = i_start; i < i_end; i++)
		for (int j = j_start; j < j_end; j++)
			sciddicaTSimulationInit(i, j, r, c, Sz, Sh);

	double * d_Sz, *d_Sh, *d_Sf;
	int size = r*c*sizeof(double);
	hipMalloc((void**) &d_Sz, size);
	hipMalloc((void**) &d_Sh, size);
	hipMalloc((void**) &d_Sf, size*4);

	hipMemcpy(d_Sz, Sz, size, hipMemcpyHostToDevice);
	hipMemcpy(d_Sh, Sh, size, hipMemcpyHostToDevice);
	hipMemcpy(d_Sf, Sf, size*4, hipMemcpyHostToDevice);

	// Apply the init kernel (elementary process) to the whole domain grid (cellular space)
	dim3 block_size(bs_x, bs_y, 1);
	dim3 number_of_blocks(ceil((r-1) / (float)block_size.x), ceil((c-1) / (float)block_size.y), 1);

	printf("r %d, c %d\n", r, c);
	printf("bs_x %d, bs_y %d\n", bs_x, bs_y);
	printf("nb_x %d, nb_y %d\n", number_of_blocks.x, number_of_blocks.y);

	util::Timer cl_timer;
	// simulation loop
	for (int s = 0; s < steps; ++s)
	{
		// Apply the resetFlow kernel to the whole domain
		sciddicaTResetFlows<<<number_of_blocks, block_size>>>(r, c, nodata, d_Sf);
		hipError_t err = hipGetLastError();
		if ( err != hipSuccess )
		{
			printf("CUDA Error: %s\n", hipGetErrorString(err));
			// Possibly: exit(-1) if program cannot continue....
		}
		// Apply the FlowComputation kernel to the whole domain
		sciddicaTFlowsComputation<<<number_of_blocks, block_size>>>(r, c, nodata, d_Sz, d_Sh, d_Sf, p_r, p_epsilon);
		err = hipGetLastError();
		if ( err != hipSuccess )
		{
			printf("CUDA Error: %s\n", hipGetErrorString(err));
			// Possibly: exit(-1) if program cannot continue....
		}
		// Apply the WidthUpdate mass balance kernel to the whole domain
		sciddicaTWidthUpdate<<<number_of_blocks, block_size>>>(r, c, nodata, d_Sz, d_Sh, d_Sf);
		err = hipGetLastError();
		if ( err != hipSuccess )
		{
			printf("CUDA Error: %s\n", hipGetErrorString(err));
			// Possibly: exit(-1) if program cannot continue....
		}
	}
	hipDeviceSynchronize();

	double cl_time = static_cast<double>(cl_timer.getTimeMilliseconds()) / 1000.0;
	printf("Elapsed time: %lf [s]\n", cl_time);

	hipMemcpy(Sh, d_Sh, size, hipMemcpyDeviceToHost);
	saveGrid2Dr(Sh, r, c, argv[OUTPUT_PATH_ID]);// Save Sh to file

	printf("Releasing memory....\n");
	delete[] Sz;
	delete[] Sh;
	delete[] Sf;
	hipFree(d_Sz);
	hipFree(d_Sh);
	hipFree(d_Sf);
	return 0;
}
